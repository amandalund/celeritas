#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2021 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file LDemoKernel.cu
//---------------------------------------------------------------------------//
#include "LDemoKernel.hh"

#include <thrust/device_ptr.h>
#include <thrust/transform_reduce.h>
#include "base/KernelParamCalculator.cuda.hh"
#include "base/StackAllocator.hh"
#include "physics/base/CutoffView.hh"
#include "random/RngEngine.hh"
#include "sim/SimTrackView.hh"
#include "KernelUtils.hh"

using namespace celeritas;

namespace demo_loop
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Whether the track is alive.
 */
struct alive
{
    __device__ size_type operator()(const SimTrackState& sim) const
    {
        return sim.alive ? 1 : 0;
    }
};

//---------------------------------------------------------------------------//
/*!
 * Sample mean free path and calculate physics step limits.
 */
__global__ void
pre_step_kernel(ParamsDeviceRef const params, StateDeviceRef const states)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= states.size())
        return;

    SimTrackView sim(states.sim, tid);
    if (!sim.alive())
        return;

    ParticleTrackView particle(params.particles, states.particles, tid);
    GeoTrackView      geo(params.geometry, states.geometry, tid);
    GeoMaterialView   geo_mat(params.geo_mats, geo.volume_id());
    MaterialTrackView mat(params.materials, states.materials, tid);
    PhysicsTrackView  phys(params.physics,
                          states.physics,
                          particle.particle_id(),
                          geo_mat.material_id(),
                          tid);
    RngEngine         rng(states.rng, ThreadId(tid));

    // Sample mfp and calculate minimum step (interaction or step-limited)
    demo_loop::calc_step_limits(mat, particle, phys, sim, rng);
}

//---------------------------------------------------------------------------//
/*!
 * Propagate and process physical changes to the track along the step and
 * select the process/model for discrete interaction.
 */
__global__ void along_and_post_step_kernel(ParamsDeviceRef const params,
                                           StateDeviceRef const  states)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= states.size())
        return;

    SimTrackView sim(states.sim, tid);
    if (!sim.alive())
        return;

    ParticleTrackView particle(params.particles, states.particles, tid);
    GeoTrackView      geo(params.geometry, states.geometry, tid);
    GeoMaterialView   geo_mat(params.geo_mats, geo.volume_id());
    PhysicsTrackView  phys(params.physics,
                          states.physics,
                          particle.particle_id(),
                          geo_mat.material_id(),
                          tid);
    RngEngine         rng(states.rng, ThreadId(tid));

    // Move particle and determine the actual distance traveled
    real_type step = demo_loop::propagate(geo, phys);

    // Calculate energy loss over the step length
    auto eloss = calc_energy_loss(particle, phys, step);
    states.energy_deposition[tid] += eloss.value();

    // The particle entered a new volume before reaching the interaction
    if (step < phys.step_length())
    {
        states.interactions[tid]
            = Interaction::from_boundary(particle.energy(), geo.dir());
    }

    // TODO: is this right??
    // Kill the track if it's outside the valid geometry region
    if (geo.is_outside())
        sim.alive(false);

    // Select the model for the discrete process
    demo_loop::select_discrete_model(particle, phys, rng, step, eloss);
}

//---------------------------------------------------------------------------//
/*!
 * Postprocessing of secondaries and interaction results.
 */
__global__ void process_interactions_kernel(ParamsDeviceRef const params,
                                            StateDeviceRef const  states)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= states.size())
        return;

    SimTrackView sim(states.sim, tid);
    if (!sim.alive())
        return;

    ParticleTrackView particle(params.particles, states.particles, tid);
    GeoTrackView      geo(params.geometry, states.geometry, tid);
    MaterialTrackView mat(params.materials, states.materials, tid);
    GeoMaterialView   geo_mat(params.geo_mats, geo.volume_id());
    PhysicsTrackView  phys(params.physics,
                          states.physics,
                          particle.particle_id(),
                          geo_mat.material_id(),
                          tid);
    CutoffView        cutoffs(params.cutoffs, mat.material_id());

    // Update the track state from the interaction
    const Interaction& result = states.interactions[tid];
    if (action_killed(result.action))
    {
        sim.alive(false);
    }
    else if (!action_unchanged(result.action)
             && !action_crossed_boundary(result.action))
    {
        particle.energy(result.energy);
        geo.set_dir(result.direction);
    }

    // Deposit energy from interaction
    states.energy_deposition[tid] += result.energy_deposition.value();

    // Kill secondaries with energy below the production threshold and deposit
    // their energy
    for (auto& secondary : result.secondaries)
    {
        if (secondary.energy < cutoffs.energy(secondary.particle_id))
        {
            states.energy_deposition[tid] += secondary.energy.value();
            secondary = {};
        }
    }

    // Reset the physics state if a discrete interaction occured
    if (phys.model_id())
        phys = {};
}

//---------------------------------------------------------------------------//
/*!
 * Clear secondaries.
 */
__global__ void
cleanup_kernel(ParamsDeviceRef const params, StateDeviceRef const states)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    StackAllocator<Secondary> allocate_secondaries(states.secondaries);

    if (tid.get() == 0)
    {
        allocate_secondaries.clear();
    }
}

} // namespace

//---------------------------------------------------------------------------//
// KERNEL INTERFACES
//---------------------------------------------------------------------------//
#define CDL_LAUNCH_KERNEL(NAME, THREADS, ARGS...)                   \
    do                                                              \
    {                                                               \
        static const ::celeritas::KernelParamCalculator NAME##_ckp( \
            NAME##_kernel, #NAME);                                  \
        auto kp = NAME##_ckp(THREADS);                              \
                                                                    \
        NAME##_kernel<<<kp.grid_size, kp.block_size>>>(ARGS);       \
        CELER_CUDA_CHECK_ERROR();                                   \
    } while (0)

//---------------------------------------------------------------------------//
/*!
 * Get minimum step length from interactions.
 */
void pre_step(const ParamsDeviceRef& params, const StateDeviceRef& states)
{
    CDL_LAUNCH_KERNEL(pre_step, states.size(), params, states);
}

//---------------------------------------------------------------------------//
/*!
 * Propogation, slowing down, and discrete model selection.
 */
void along_and_post_step(const ParamsDeviceRef& params,
                         const StateDeviceRef&  states)
{
    CDL_LAUNCH_KERNEL(along_and_post_step, states.size(), params, states);
}

//---------------------------------------------------------------------------//
/*!
 * Postprocessing of secondaries and interaction results.
 */
void process_interactions(const ParamsDeviceRef& params,
                          const StateDeviceRef&  states)
{
    CDL_LAUNCH_KERNEL(process_interactions, states.size(), params, states);
}

//---------------------------------------------------------------------------//
/*!
 * Get the number of active tracks.
 */
size_type reduce_alive(const StateDeviceRef& states)
{
    auto sim_states = states.sim.state[AllItems<SimTrackState>{}].data();
    return thrust::transform_reduce(
        thrust::device_pointer_cast(sim_states),
        thrust::device_pointer_cast(sim_states) + states.size(),
        alive(),
        0,
        thrust::plus<size_type>());
}

//---------------------------------------------------------------------------//
/*!
 * Clear secondaries.
 */
void cleanup(const ParamsDeviceRef& params, const StateDeviceRef& states)
{
    CDL_LAUNCH_KERNEL(cleanup, 1, params, states);
}

//---------------------------------------------------------------------------//
} // namespace demo_loop
