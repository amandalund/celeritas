#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file InitializeTracks.cu
//---------------------------------------------------------------------------//
#include "InitializeTracks.hh"

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/remove.h>
#include <thrust/scan.h>
#include <vector>
#include "base/Atomics.hh"
#include "base/DeviceVector.hh"

namespace celeritas
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Initialize the track states on device. The track initializers are created
 * from either primary particles or secondaries. The new tracks are inserted
 * into empty slots (vacancies) in the track vector.
 */
__global__ void init_tracks_kernel(const StatePointers            states,
                                   const ParamPointers            params,
                                   const TrackInitializerPointers inits)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < inits.vacancies.size())
    {
        // Get the track initializer, starting from the back of the vector
        const TrackInitializer& init
            = inits.initializers[inits.initializers.size() - thread_id - 1];

        // Index of the empty slot to create the new track in
        ThreadId slot_id(inits.vacancies[thread_id]);

        // Initialize the simulation state
        SimTrackView sim(states.sim, slot_id);
        sim = init.sim;

        // Initialize the particle physics data
        ParticleTrackView particle(params.particle, states.particle, slot_id);
        particle = init.particle;

        // Copy the geometry state from the parent if possible
        GeoTrackView geo(params.geo, states.geo, slot_id);
        if (thread_id < inits.parent.size())
        {
            unsigned int parent_id
                = inits.parent[inits.parent.size() - thread_id - 1];
            GeoTrackView parent(params.geo, states.geo, ThreadId(parent_id));
            geo.copy_state(parent, init.geo.dir);
        }
        // Initialize it from the position otherwise
        else
        {
            geo = init.geo;
        }
    }
}

//---------------------------------------------------------------------------//
/*!
 * Find empty slots in the track vector and count the number of secondaries
 * that survived cutoffs for each interaction. If the track is dead and
 * produced secondaries, fill the empty track slot with one of the secondaries.
 */
__global__ void locate_alive_kernel(const StatePointers            states,
                                    const ParamPointers            params,
                                    const TrackInitializerPointers inits)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < states.size())
    {
        // Secondary to copy to the parent's track slot if the parent has died
        Secondary firstborn = Secondary::from_cutoff();

        // Count how many secondaries survived cutoffs for each track
        inits.secondary_counts[thread_id] = 0;
        for (const auto& secondary : states.interactions[thread_id].secondaries)
        {
            if (secondary)
            {
                if (!firstborn)
                {
                    firstborn = secondary;
                }
                ++inits.secondary_counts[thread_id];
            }
        }

        SimTrackView sim(states.sim, ThreadId(thread_id));

        // The track is alive: mark this track slot as active
        if (sim.alive())
        {
            inits.vacancies[thread_id] = flag_alive();
        }
        // The track is dead and produced secondaries: fill the empty track
        // slot with the first secondary and mark the track slot as active
        else if (firstborn)
        {
            // TODO: calculate the correct track ID for the secondary
            // Initialize the simulation state
            sim = {TrackId{}, sim.track_id(), sim.event_id(), true};

            // Initialize the particle state from the secondary
            ParticleTrackView particle(
                params.particle, states.particle, ThreadId(thread_id));
            particle = {firstborn.def_id, firstborn.energy};

            // Keep the parent's geometry state
            GeoTrackView geo(params.geo, states.geo, ThreadId(thread_id));
            geo.copy_state(geo, firstborn.direction);

            // Mark the secondary as processed and the track as active
            --inits.secondary_counts[thread_id];
            firstborn                  = Secondary::from_cutoff();
            inits.vacancies[thread_id] = flag_alive();
        }
        // The track is dead and did not produce secondaries: store the index
        // so it can be used later to initialize a new track
        else
        {
            inits.vacancies[thread_id] = thread_id;
        }
    }
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers on device from primary particles.
 */
__global__ void process_primaries_kernel(const span<const Primary> primaries,
                                         const TrackInitializerPointers inits)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < primaries.size())
    {
        TrackInitializer& init
            = inits.initializers[inits.initializers.size() + thread_id];

        // Construct a track initializer from a primary particle
        init = primaries[thread_id];
    }
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers on device from secondary particles.
 */
__global__ void process_secondaries_kernel(const StatePointers states,
                                           const ParamPointers params,
                                           const TrackInitializerPointers inits)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < states.size())
    {
        // Construct the state accessors
        GeoTrackView geo(params.geo, states.geo, ThreadId(thread_id));
        SimTrackView sim(states.sim, ThreadId(thread_id));

        // Offset in the vector of track initializers
        size_type offset_id = inits.secondary_counts[thread_id];

        for (const auto& secondary : states.interactions[thread_id].secondaries)
        {
            // If the secondary survived cutoffs
            if (secondary)
            {
                TrackInitializer& init
                    = inits.initializers[inits.initializers.size() + offset_id];

                // Store the thread ID of the secondary's parent
                inits.parent[offset_id] = thread_id;

                // Calculate the track ID of the secondary
                unsigned int track_id = 1 + inits.track_count + offset_id++;

                // Construct a track initializer from a secondary
                init.sim.track_id    = TrackId{track_id};
                init.sim.parent_id   = sim.track_id();
                init.sim.event_id    = sim.event_id();
                init.sim.alive       = true;
                init.particle.def_id = secondary.def_id;
                init.particle.energy = secondary.energy;
                init.geo.dir         = secondary.direction;
                init.geo.pos         = geo.pos();
            }
        }
    }
}

//---------------------------------------------------------------------------//
// KERNEL INTERFACE
//---------------------------------------------------------------------------//
/*!
 * Initialize the track states on device.
 */
void init_tracks(StatePointers            states,
                 ParamPointers            params,
                 TrackInitializerPointers inits)
{
    // Initialize tracks on device
    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(inits.vacancies.size());
    init_tracks_kernel<<<lparams.grid_size, lparams.block_size>>>(
        states, params, inits);

    CELER_CUDA_CALL(hipDeviceSynchronize());
}

//---------------------------------------------------------------------------//
/*!
 * Find empty slots in the vector of tracks and count the number of secondaries
 * that survived cutoffs for each interaction.
 */
void locate_alive(StatePointers            states,
                  ParamPointers            params,
                  TrackInitializerPointers inits)
{
    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(states.size());
    locate_alive_kernel<<<lparams.grid_size, lparams.block_size>>>(
        states, params, inits);

    CELER_CUDA_CALL(hipDeviceSynchronize());
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers from primary particles.
 */
void process_primaries(span<const Primary>      primaries,
                       TrackInitializerPointers inits)
{
    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(primaries.size());
    process_primaries_kernel<<<lparams.grid_size, lparams.block_size>>>(
        primaries, inits);

    CELER_CUDA_CALL(hipDeviceSynchronize());
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers from secondary particles.
 */
void process_secondaries(StatePointers            states,
                         ParamPointers            params,
                         TrackInitializerPointers inits)
{
    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(states.size());
    process_secondaries_kernel<<<lparams.grid_size, lparams.block_size>>>(
        states, params, inits);

    CELER_CUDA_CALL(hipDeviceSynchronize());
}

//---------------------------------------------------------------------------//
/*!
 * Remove all elements in the vacancy vector that were flagged as active
 * tracks.
 */
size_type remove_if_alive(span<size_type> vacancies)
{
    thrust::device_ptr<size_type> end = thrust::remove_if(
        thrust::device_pointer_cast(vacancies.data()),
        thrust::device_pointer_cast(vacancies.data() + vacancies.size()),
        alive(flag_alive()));

    CELER_CUDA_CALL(hipDeviceSynchronize());

    // New size of the vacancy vector
    size_type result = thrust::raw_pointer_cast(end) - vacancies.data();
    return result;
}

//---------------------------------------------------------------------------//
/*!
 * Sum the total number of surviving secondaries.
 */
size_type reduce_counts(span<size_type> counts)
{
    size_type result = thrust::reduce(
        thrust::device_pointer_cast(counts.data()),
        thrust::device_pointer_cast(counts.data()) + counts.size(),
        size_type(0),
        thrust::plus<size_type>());

    CELER_CUDA_CALL(hipDeviceSynchronize());
    return result;
}

//---------------------------------------------------------------------------//
/*!
 * Calculate the exclusive prefix sum of the number of surviving secondaries
 * from each interaction.
 */
void exclusive_scan_counts(span<size_type> counts)
{
    thrust::exclusive_scan(
        thrust::device_pointer_cast(counts.data()),
        thrust::device_pointer_cast(counts.data()) + counts.size(),
        counts.data(),
        size_type(0));

    CELER_CUDA_CALL(hipDeviceSynchronize());
}

//---------------------------------------------------------------------------//
} // namespace celeritas
